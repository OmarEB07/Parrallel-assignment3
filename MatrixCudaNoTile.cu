#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrixMultiplication(float *a, float *b, float *c, int M, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < M && j < N) {
        float sum = 0.0;
        for (int k = 0; k < N; k++) {
            sum += a[i * N + k] * b[k * N + j];
        }
        c[i * N + j] = sum;
    }
}

int main()
{
    int M = 1000;
    int N = 500;

    float *a, *b, *c;
    float *d_a, *d_b, *d_c;
    int size_a = M * N * sizeof(float);
    int size_b = N * N * sizeof(float);
    int size_c = M * N * sizeof(float);

    // Allocate memory on host
    a = (float*) malloc(size_a);
    b = (float*) malloc(size_b);
    c = (float*) malloc(size_c);

    // Initialize input matrices
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            a[i * N + j] = i + j;
        }
    }
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            b[i * N + j] = i - j;
        }
    }

    // Allocate memory on device
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    // Copy input matrices from host to device
    hipMemcpy(d_a, a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size_b, hipMemcpyHostToDevice);

    // Define grid and block sizes
    dim3 dimGrid((M + 15) / 16, (N + 15) / 16, 1);
    dim3 dimBlock(16, 16, 1);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record start event
    hipEventRecord(start);

    // Call kernel function
    matrixMultiplication<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, M, N);

    // Record stop event
    hipEventRecord(stop);

    // Wait for the completion of all device operations
    hipDeviceSynchronize();

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Copy output matrix from device to host
    hipMemcpy(c, d_c, size_c, hipMemcpyDeviceToHost);

   // Print output matrix
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", c[i * N + j]);
        }
        printf("\n");
    }


    // Print execution time
    printf("Execution time: %.2f ms\n", milliseconds);

    // Free memory
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
